#include "hip/hip_runtime.h"
//
// Created by ron on 8/7/20.
//

#include "og.h"
namespace titan {
OccupancyGrid::OccupancyGrid() {
    arrayptr = nullptr;
    ref_count = 0;
    // initialize with empty unordered sets
    for (int i = 0; i < size_x*size_y*size_b; i++){
        grid.push_back(std::unordered_set<Mass *>());
    }
}

    void OccupancyGrid::decrementRefCount() { ref_count--; }
    void OccupancyGrid::update(Mass **masses, int nr_masses) {
        for (int i = 0; i < nr_masses; ++i){
        }

    }

    void OccupancyGrid::insert(Mass * m) {
        Vec tmp;
        int x_val, y_val;
        Vec scale = Vec(size_x, size_y, 1);
        tmp = m->pos/scale;
        x_val = int(std::floor(tmp[0])+center_x);
        y_val = int(std::floor(tmp[1])+center_x);
        x_val = (x_val > (size_x-1) ? size_x-1 : x_val);
        y_val = (y_val > (size_y-1) ? size_y-1 : y_val);

    }

    bool OccupancyGrid::isInCell(int x, int y) {
        return false;
    }
    int OccupancyGrid::idx2DTo1D(int x, int y) {
        int idx_1D = x*size_x+y;
    }

}