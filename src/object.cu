#include "hip/hip_runtime.h"
//
// Created by Jacob Austin on 5/21/18.
// object.cu defines constraint objects like planes and balls that allow the users
// to enforce limitations on movements of objects within the scene.
// Generally, an object defines the applyForce method that determines whether to apply a force
// to a mass, for example a normal force pushing the mass out of a constaint object or
// a frictional force.

#include "object.h"
#include <cmath>
#include "sim.h"

#ifdef GRAPHICS
#define GLM_FORCE_PURE
#define GLM_ENABLE_EXPERIMENTAL
#include <glm/gtx/quaternion.hpp> // for rotation
#endif

namespace titan {

#ifdef GRAPHICS
const Vec RED(1.0, 0.2, 0.2);
const Vec GREEN(0.2, 1.0, 0.2);
const Vec BLUE(0.2, 0.2, 1.0);
const Vec PURPLE(0.5, 0.2, 0.5);

#endif

__device__ const double NORMAL = 20000; // normal force coefficient for contact constaints

#ifdef CONSTRAINTS
void Container::addConstraint(CONSTRAINT_TYPE type, const Vec & v, double d) {
    for (Mass * m : masses) {
        m -> addConstraint(type, v, d);
    }
}

void Container::clearConstraints() {
    for (Mass * m : masses) {
        m -> clearConstraints();
    }
}

#endif

CUDA_CALLABLE_MEMBER CudaBall::CudaBall(const Vec & center, double radius) {
    _center = center;
    _radius = radius;
}

CUDA_CALLABLE_MEMBER CudaBall::CudaBall(const Ball & b) {
    _center = b._center;
    _radius = b._radius;
}

CUDA_CALLABLE_MEMBER void CudaBall::applyForce(CUDA_MASS * m) {
    double dist = (m -> pos - _center).norm();
    m -> force += (dist <= _radius) ? NORMAL * (m -> pos - _center) / dist : Vec(0, 0, 0);
}

CUDA_CALLABLE_MEMBER CudaContactPlane::CudaContactPlane(const Vec & normal, double offset) {
    _normal = normal / normal.norm();
    _offset = offset;
    _FRICTION_S = 0.0;
    _FRICTION_K = 0.0;
}

CudaContactPlane::CudaContactPlane(const ContactPlane & p) {
    _normal = p._normal;
    _offset = p._offset;

    _FRICTION_S = p._FRICTION_S;
    _FRICTION_K = p._FRICTION_K;
}

CUDA_CALLABLE_MEMBER void CudaContactPlane::applyForce(CUDA_MASS * m) {
    //    m -> force += (disp < 0) ? - disp * NORMAL * _normal : 0 * _normal; // TODO fix this for the host

    double disp = dot(m -> pos, _normal) - _offset; // displacement into the plane
    Vec f_normal = dot(m -> force, _normal) * _normal; // normal force

    if (disp < 0 && (_FRICTION_S > 0 || _FRICTION_K > 0)) { // if inside the plane
        Vec v_perp = m -> vel - dot(m -> vel, _normal) * _normal; // perpendicular velocity
        double v_norm = v_perp.norm();

        if (v_norm > 1e-16) { // kinetic friction domain
            double friction_mag = _FRICTION_K * f_normal.norm();
            m->force -= v_perp * friction_mag / v_norm;
        } else { // static friction
            Vec f_perp = m -> force - f_normal; // perpendicular force
	        if (_FRICTION_S * f_normal.norm() > f_perp.norm()) {
                m -> force -= f_perp;
	        } // else { // kinetic domain again
            //     double friction_mag = _FRICTION_K * f_normal.norm();
            //     m->force -= v_perp * friction_mag / v_norm;
	        // }
        }
    }

    // now apply the offset force to push the object out of the plane.
    // if (disp < 0) {
    //     m -> pos[2] = 0;
    //     m -> vel = m -> vel - 2 * dot(m -> vel, _normal) * _normal;
    //     m -> force -= f_normal;
    // }

    Vec contact = (disp < 0) ? - disp * NORMAL * _normal : 0 * _normal; // displacement force
    m -> force += contact;
}

CUDA_CALLABLE_MEMBER CudaConstraintPlane::CudaConstraintPlane(const Vec & normal, double friction) {
    assert(normal.norm() != 0.0);

    _normal = normal / normal.norm();
    _friction = friction;
}

CUDA_CALLABLE_MEMBER void CudaConstraintPlane::applyForce(CUDA_MASS * m) {
    double normal_force = dot(m -> force, _normal);
    m -> force += - _normal * normal_force; // constraint force
    double v_norm = m -> vel.norm();

    if (v_norm >= 1e-16) {
        m -> vel += - _normal * dot(m -> vel, _normal); // constraint velocity
        m -> force += - _friction * normal_force * m -> vel / v_norm; // apply friction force
    }
}

CUDA_CALLABLE_MEMBER CudaDirection::CudaDirection(const Vec & tangent, double friction) {
    assert(tangent.norm() != 0.0);

    _tangent = tangent / tangent.norm();
    _friction = friction;
}

CUDA_CALLABLE_MEMBER void CudaDirection::applyForce(CUDA_MASS * m) {
    Vec normal_force = m -> force - dot(m -> force, _tangent) * _tangent;
    m -> force += - normal_force;

    if (m -> vel.norm() >= 1e-16) {
        m -> vel = _tangent * dot(m -> vel, _tangent);
        m -> force += - normal_force.norm() * _friction * _tangent;
    }
}

void Container::setMassValues(double m) { // set masses for all Mass objects
    for (Mass * mass : masses) {
        mass -> m += m;
    }
}

void Container::setSpringConstants(double k) {
    for (Spring * spring : springs) {
        spring -> _k = k;
    }
}

void Container::setRestLengths(double len) { // set masses for all Mass objects
    for (Spring * spring : springs) {
        spring -> _rest = len;
    }
}

void Container::add(Mass * m) {
    masses.push_back(m);
}

void Container::add(Spring * s) {
    springs.push_back(s);
}

void Container::add(Container * c) {
    for (Mass * m : c -> masses) {
        masses.push_back(m);
    }

    for (Spring * s : c -> springs) {
        springs.push_back(s);
    }
}

Cube::Cube(const Vec & center, double side_length) {
    _center = center;
    _side_length = side_length;

    for (int i = 0; i < 8; i++) {
        masses.push_back(new Mass(side_length * (Vec(i & 1, (i >> 1) & 1, (i >> 2) & 1) - Vec(0.5, 0.5, 0.5)) + center));
    }

    for (int i = 0; i < 8; i++) { // add the appropriate springs
        for (int j = i + 1; j < 8; j++) {
            springs.push_back(new Spring(masses[i], masses[j]));
        }
    }

    for (Spring * s : springs) {
        s -> setRestLength((s -> _right -> pos - s -> _left -> pos).norm());
    }
}

void Container::translate(const Vec & displ) {
    for (Mass * m : masses) {
        m -> pos += displ;
    }
}

void Container::rotate(const Vec & axis, double angle) {
    Vec com(0, 0, 0);

    double total_mass = 0;

    for (Mass * m : masses) {
        com += m -> m * m -> pos;
        total_mass += m -> m;
    }

    com = com / total_mass; // center of mass as centroid
    Vec temp_axis = axis / axis.norm();

    for (Mass * m : masses) {
        Vec temp = m -> pos - com; // subtract off center of mass
        Vec y = temp - dot(temp, temp_axis) * temp_axis; // project onto the given axis and find offset (y coordinate)

        if (y.norm() < 0.0001) { // if on the axis, don't do anything
            continue;
        }

        Vec planar(-sin(angle) * y.norm(), cos(angle) * y.norm(), 0); // coordinate in xy space
        Vec spatial = planar[0] * cross(temp_axis, y / y.norm()) + y / y.norm() * planar[1] + dot(temp, temp_axis) * temp_axis + com; // return to 3D space, then to COM space, then to absolute space

        m -> pos = spatial; // update position
    }
}

Lattice::Lattice(const Vec & center, const Vec & dims, int nx, int ny, int nz) {
    _center = center;
    _dims = dims;
    this -> nx = nx;
    this -> ny = ny;
    this -> nz = nz;

    for (int i = 0; i < nx; i++) {
        for (int j = 0; j < ny; j++) {
            for (int k = 0; k < nz; k++) {
                masses.push_back(new Mass(Vec((nx > 1) ? (double) i / (nx - 1.0) - 0.5 : 0, (ny > 1) ? j / (ny - 1.0) - 0.5 : 0, (nz > 1) ? k / (nz - 1.0) - 0.5 : 0) * dims + center));
            }
        }
    }

    for (int i = 0; i < nx; i++) {
        for (int j = 0; j < ny; j++) {
            for (int k = 0; k < nz; k++) {
                for (int l = 0; l < ((i != nx - 1) ? 2 : 1); l++) {
                    for (int m = 0; m < ((j != ny - 1) ? 2 : 1); m++) {
                        for (int n = 0; n < ((k != nz - 1) ? 2 : 1); n++) {
                            if (l != 0 || m != 0 || n != 0) {
                                springs.push_back(new Spring(masses[k + j * nz + i * ny * nz],
                                                             masses[(k + n) + (j + m) * nz + (i + l) * ny * nz]));
                            }
                        }
                    }
                }

                if (k != nz - 1) {
                    if (j != ny - 1) {
                        springs.push_back(new Spring(masses[(k + 1) + j * nz + i * ny * nz], // get the full triangle
                                                     masses[k + (j + 1) * nz + i * ny * nz]));
                    }

                    if (i != nx - 1) {
                        springs.push_back(new Spring(masses[(k + 1) + j * nz + i * ny * nz],
                                                     masses[k + j * nz + (i + 1) * ny * nz]));
                    }

                    if (j != ny - 1 && i != nx - 1) {
                        springs.push_back(new Spring(masses[(k + 1) + j * nz + i * ny * nz],
                                                     masses[k + (j + 1) * nz + (i + 1) * ny * nz]));
                        springs.push_back(new Spring(masses[(k + 1) + j * nz + (i + 1) * ny * nz],
                                                     masses[k + (j + 1) * nz + i * ny * nz]));
                        springs.push_back(new Spring(masses[(k + 1) + (j + 1) * nz + i * ny * nz],
                                                     masses[k + j * nz + (i + 1) * ny * nz]));
                    }
                }

                if (j != ny - 1 && i != nx - 1) {
                    springs.push_back(new Spring(masses[k + (j + 1) * nz + i * ny * nz],
                                                 masses[k + j * nz + (i + 1) * ny * nz]));
                }
            }
        }
    }

    for (Spring * s : springs) {
        s -> setRestLength((s -> _right -> pos - s -> _left -> pos).norm());
    }
}

#ifdef CONSTRAINTS
Beam::Beam(const Vec & center, const Vec & dims, int nx, int ny, int nz) {
    _center = center;
    _dims = dims;
    this -> nx = nx;
    this -> ny = ny;
    this -> nz = nz;

    for (int i = 0; i < nx; i++) {
        for (int j = 0; j < ny; j++) {
            for (int k = 0; k < nz; k++) {
	            masses.push_back(new Mass(Vec((nx > 1) ? (double) i / (nx - 1.0) - 0.5 : 0, (ny > 1) ? j / (ny - 1.0) - 0.5 : 0, (nz > 1) ? k / (nz - 1.0) - 0.5 : 0) * dims + center));
                if (i == 0) {
                    masses[masses.size() - 1] -> constraints.fixed = true;
                }
            }
        }
    }

    for (int i = 0; i < nx; i++) {
        for (int j = 0; j < ny; j++) {
            for (int k = 0; k < nz; k++) {
                for (int l = 0; l < ((i != nx - 1) ? 2 : 1); l++) {
                    for (int m = 0; m < ((j != ny - 1) ? 2 : 1); m++) {
                        for (int n = 0; n < ((k != nz - 1) ? 2 : 1); n++) {
                            if (l != 0 || m != 0 || n != 0) {
                                springs.push_back(new Spring(masses[k + j * nz + i * ny * nz],
                                                             masses[(k + n) + (j + m) * nz + (i + l) * ny * nz]));
                            }
                        }
                    }
                }

                if (k != nz - 1) {
                    if (j != ny - 1) {
                        springs.push_back(new Spring(masses[(k + 1) + j * nz + i * ny * nz], // get the full triangle
                                                     masses[k + (j + 1) * nz + i * ny * nz]));
                    }

                    if (i != nx - 1) {
                        springs.push_back(new Spring(masses[(k + 1) + j * nz + i * ny * nz],
                                                     masses[k + j * nz + (i + 1) * ny * nz]));
                    }

                    if (j != ny - 1 && i != nx - 1) {
                        springs.push_back(new Spring(masses[(k + 1) + j * nz + i * ny * nz],
                                                     masses[k + (j + 1) * nz + (i + 1) * ny * nz]));
                        springs.push_back(new Spring(masses[(k + 1) + j * nz + (i + 1) * ny * nz],
                                                     masses[k + (j + 1) * nz + i * ny * nz]));
                        springs.push_back(new Spring(masses[(k + 1) + (j + 1) * nz + i * ny * nz],
                                                     masses[k + j * nz + (i + 1) * ny * nz]));
                    }
                }

                if (j != ny - 1 && i != nx - 1) {
                    springs.push_back(new Spring(masses[k + (j + 1) * nz + i * ny * nz],
                                                 masses[k + j * nz + (i + 1) * ny * nz]));
                }
            }
        }
    }

    for (Spring * s : springs) {
        s -> setRestLength((s -> _right -> pos - s -> _left -> pos).norm());
    }
}
#endif
/*
 * Robot Link Class Functions
 */
    RobotLink::RobotLink(const Vec &pos1, const Vec &pos2, double mass, double max_exp_length, double min_exp_length,
                         double expansion_rate, double k, double magnetic_force, double radius) {
        max_length = max_exp_length; // expanded link length (magnet center to magnet center)
        min_length = min_exp_length; // contracted length of link
        k_link = k; // link stiffness
        max_mag_force = magnetic_force; // magnetic force of connector
        // shell stiffness
        ml = new Mass(pos1, mass, false, radius, 5000.0, max_mag_force, 1.0);
        mr = new Mass(pos2, mass, false, radius, 5000.0, max_mag_force, 1.0);
        s = new Spring(ml, mr, k_link, min_length, PASSIVE_SOFT, 0.0, max_length,
                min_length, expansion_rate);
        s->_rest = min_length;
        // add masses and springs to respective vectors
        masses.push_back(ml);
        masses.push_back(mr);
        springs.push_back(s);
    }
    /*
     * ToDo: Implement class functions
     */
    bool RobotLink::expand() {
        if (max_length <= s->_rest){
            s->_type = PASSIVE_SOFT;
            return false;
        } else {
            s->_type = ACTUATED_EXPAND;
            this->attach(); // the expanding link is always in attachment mode
            return true;
        }
    }

    bool RobotLink::contract() {
        if (min_length >= s->_rest){
            s->_type = PASSIVE_SOFT;
            return false;
        } else {
            s->_type = ACTUATED_EXPAND;
            return true;
        }
    }

    // removes magnet force from masses
    bool RobotLink::detach() {
        if (!this->contract()){
            if (ml->isMagnetic()){
                ml->max_mag_force = 0.0;
            }
            if (mr->isMagnetic()){
                mr->max_mag_force = 0.0;
            }
            return true;
        }
        return false;
    }
    // adds magnet force to masses
    bool RobotLink::attach() {
        if (!ml->isMagnetic()){
            ml->max_mag_force = max_mag_force;
        }
        if (!mr->isMagnetic()){
            mr->max_mag_force = max_mag_force;
        }
        return false;
    }

    void RobotLink::setExpansionRate(double exp_rate) {
        this->exp_rate = exp_rate;
        s->_rate = exp_rate;
    }

    void RobotLink::setRobotMass(double mass) {
        ml->m = mass/2;
        mr->m = mass/2;
    }

    void RobotLink::setColor(Vec c) {
        ml->color = c;
        mr->color = c;
    }

    void RobotLink::setStiffness(double k) {
        k_link = k;
        s->_k = k;
    }

// Robot::Robot(const Vec & center, const cppn& encoding, double side_length,  double omega, double k_soft, double k_stiff){
//     _center = center;
//     _side_length = side_length;
//     _omega = omega;
//     _k_soft = k_soft;
//     _k_stiff = k_stiff;
//     _encoding = encoding;
    
//     int RobotDim = encoding.size(); // number of cubes per side
//     Vec dims(side_length,side_length,side_length);
//     // keep trace of number of cubes that each mass is connected to 
//     std::vector<std::vector<std::vector<int>>> mass_conn(RobotDim+1, std::vector<std::vector<int>>(RobotDim+1,std::vector<int>(RobotDim+1,0)));
    
//     std::vector<std::vector<std::vector<Mass *>>> _masses(RobotDim+1, std::vector<std::vector<Mass *>>(RobotDim+1,std::vector<Mass *>(RobotDim+1,nullptr)));
  
//     // store number of cubes that should be connected to each mass
//     for (int i = 0; i < RobotDim+1; i++) {
//       for (int j = 0; j < RobotDim+1; j++) {
// 	for (int k = 0; k < RobotDim+1; k++) {
// 	  // if index mode RobotDim+1 is 0, then it is on the edge
// 	  int i_edge = (i % (RobotDim)) ? 0:1; 
// 	  int j_edge = (j % (RobotDim)) ? 0:1;
// 	  int k_edge = (k % (RobotDim)) ? 0:1;

	
// 	  if (i_edge + j_edge + k_edge ==0){
// 	    mass_conn[i][j][k] = 8; //corner
// 	  }else if (i_edge+j_edge+k_edge ==3){
// 	    mass_conn[i][j][k] = 1; //corner
// 	  }else if (i_edge+j_edge+k_edge ==2){
// 	    mass_conn[i][j][k] = 2; //edge
// 	  }else{
// 	    mass_conn[i][j][k] = 4; //surface
// 	  }	
// 	}
//       }
//     }

//     // Remove appropriate masses
//     for (int i = 0; i < RobotDim; i++) {
//       for (int j = 0; j < RobotDim; j++) {
// 	for (int k = 0; k < RobotDim; k++) {
	
// 	  int exist = encoding[i][j][k][0];

// 	  if (!exist){
// 	    // subtract connectedness of each mass for the cube
// 	    mass_conn[i][j][k] -= 1;
// 	    mass_conn[i][j][k+1] -= 1;
// 	    mass_conn[i][j+1][k] -= 1;
// 	    mass_conn[i][j+1][k+1] -= 1;
// 	    mass_conn[i+1][j][k] -= 1;
// 	    mass_conn[i+1][j][k+1] -= 1;
// 	    mass_conn[i+1][j+1][k] -= 1;
// 	    mass_conn[i+1][j+1][k+1] -= 1;
// 	  }
// 	}
//       }
//     }
  
//     // create masses
//     for (int i = 0; i < RobotDim+1; i++) {
//         for (int j = 0; j < RobotDim+1; j++) {
//             for (int k = 0; k < RobotDim + 1; k++) {
//                 if (mass_conn[i][j][k] > 0){
//                     Mass * m;
//                     if (RobotDim == 1) {
//                     m = new Mass(Vec(i-0.5, j-0.5, k-0.5) * dims + _center);
//                     } else {
//                         m = new Mass(Vec(i / (RobotDim - 1.0) - 0.5,
//                                 j / (RobotDim - 1.0) - 0.5,
//                                 k / (RobotDim - 1.0) - 0.5) * dims + _center);
//                     }

// #ifdef GRAPHICS
//                     m -> color = Vec(0,0,0);
// #endif

//                     masses.push_back(m);
//                     _masses[i][j][k] = m;
//                 }
//             }
//         }
//     }


//     // create springs
//     for (int i = 0; i < RobotDim; i++) {
//         for (int j = 0; j < RobotDim; j++) {
// 	        for (int k = 0; k < RobotDim; k++) {
	
//             int exist = encoding[i][j][k][0];

//             if (exist) {
//                 int type = encoding[i][j][k][1];
            
//                 for(int l=0; l<8; l++) {
//                     int l_x = (l<4)? 0:1;
//                     int l_y = (l<2)? 0:(l<4)?1:(l<6)?0:1;
//                     int l_z = (l%2)? 1:0;
                
//                     for (int m=l+1; m<8; m++) {
//                         int r_x = (m<4)? 0:1;
//                         int r_y = (m<2)? 0:(m<4)?1:(m<6)?0:1;
//                         int r_z = (m%2)? 1:0;

//                         Spring * spr = new Spring(_masses[i+l_x][j+l_y][k+l_z],
//                                     _masses[i+r_x][j+r_y][k+r_z]);
                        
//                         spr -> _type = type;
//                         spr -> _omega = omega;

//                         if (type==0) { // green, contract then expand
//                             spr -> _k = k_soft;

// #ifdef GRAPHICS
//                             _masses[i+l_x][j+l_y][k+l_z]->color += GREEN/16;
//                             _masses[i+r_x][j+r_y][k+r_z]->color += GREEN/16;
// #endif
//                         } else if (type==1) { // red, expand then contract
//                             spr -> _k = k_soft;
// #ifdef GRAPHICS
//                             _masses[i+l_x][j+l_y][k+l_z]->color += RED/16;
//                             _masses[i+r_x][j+r_y][k+r_z]->color += RED/16;
// #endif
                
//                         } else if (type==2) { // passive soft
//                             spr -> _k = k_soft;
// #ifdef GRAPHICS
//                             _masses[i+l_x][j+l_y][k+l_z]->color += BLUE/16;
//                             _masses[i+r_x][j+r_y][k+r_z]->color += BLUE/16;
// #endif
//                         } else { // passive stiff
//                             spr -> _k = k_stiff;
// #ifdef GRAPHICS
//                             _masses[i+l_x][j+l_y][k+l_z]->color += PURPLE/16;
//                             _masses[i+r_x][j+r_y][k+r_z]->color += PURPLE/16;
// #endif
//                         }

// 		                springs.push_back(spr);
//                         }
//                     }
//                 }
//             }
//         }
//     }

    
//     for (Spring * s : springs) {
//         s -> setRestLength((s -> _right -> pos - s -> _left -> pos).norm());
//     }
// }

#ifdef CONSTRAINTS

void Container::fix() {
    for (Mass * mass : masses) {
        mass -> constraints.fixed = true;
    }
}

LOCAL_CONSTRAINTS::LOCAL_CONSTRAINTS() {
//    constraint_plane = thrust::device_vector<CudaConstraintPlane>(1);
//    contact_plane = thrust::device_vector<CudaContactPlane>(1);
//    ball = thrust::device_vector<CudaBall>(1);
//    direction = thrust::device_vector<CudaDirection>(1);

//    contact_plane_ptr = thrust::raw_pointer_cast(contact_plane.data()); // TODO make sure this is safe
//    constraint_plane_ptr = thrust::raw_pointer_cast(constraint_plane.data());
//    ball_ptr = thrust::raw_pointer_cast(ball.data());
//    direction_ptr = thrust::raw_pointer_cast(direction.data());

    num_contact_planes = 0;
    num_constraint_planes = 0;
    num_balls = 0;
    num_directions = 0;

    drag_coefficient = 0;
    fixed = false;
}

CUDA_LOCAL_CONSTRAINTS::CUDA_LOCAL_CONSTRAINTS(LOCAL_CONSTRAINTS & c) {
    contact_plane = c.contact_plane_ptr;
    constraint_plane = c.constraint_plane_ptr;
    ball = c.ball_ptr;
    direction = c.direction_ptr;

    num_contact_planes = c.num_contact_planes;
    num_constraint_planes = c.num_constraint_planes;
    num_balls = c.num_balls;
    num_directions = c.num_directions;

    fixed = c.fixed;
    drag_coefficient = c.drag_coefficient;
}

#endif

#ifdef GRAPHICS

void Ball::normalize(GLfloat * v) {
    GLfloat norm = sqrt(pow(v[0], 2) + pow(v[1], 2) + pow(v[2],2)) / _radius;

    for (int i = 0; i < 3; i++) {
        v[i] /= norm;
    }
}

void Ball::writeTriangle(GLfloat * arr, GLfloat *v1, GLfloat *v2, GLfloat *v3) {
    for (int j = 0; j < 3; j++) {
        arr[j] = v1[j] + _center[j];
    }

    arr += 3;

    for (int j = 0; j < 3; j++) {
        arr[j] = v2[j] + _center[j];
    }

    arr += 3;

    for (int j = 0; j < 3; j++) {
        arr[j] = v3[j] + _center[j];
    }
}

void Ball::subdivide(GLfloat * arr, GLfloat *v1, GLfloat *v2, GLfloat *v3, int depth) {
    GLfloat v12[3], v23[3], v31[3];

    if (depth == 0) {
        writeTriangle(arr, v1, v2, v3);
        return;
    }

    for (int i = 0; i < 3; i++) {
        v12[i] = v1[i]+v2[i];
        v23[i] = v2[i]+v3[i];
        v31[i] = v3[i]+v1[i];
    }

    normalize(v12);
    normalize(v23);
    normalize(v31);

    subdivide(arr, v1, v12, v31, depth - 1);
    arr += 3 * 3 * (int) pow(4, depth - 1);
    subdivide(arr, v2, v23, v12, depth - 1);
    arr += 3 * 3 * (int) pow(4, depth - 1);
    subdivide(arr, v3, v31, v23, depth - 1);
    arr += 3 * 3 * (int) pow(4, depth - 1);
    subdivide(arr, v12, v23, v31, depth - 1);
}


void Ball::generateBuffers() {
    glm::vec3 color = {0.22f, 0.71f, 0.0f};

    GLfloat * vertex_data = new GLfloat[20 * 3 * 3 * (int) pow(4, depth)]; // times 4 for subdivision

    GLfloat X = (GLfloat) _radius * .525731112119133606;
    GLfloat Z = (GLfloat) _radius * .850650808352039932;

    static GLfloat vdata[12][3] = {
            {-X, 0.0, Z}, {X, 0.0, Z}, {-X, 0.0, -Z}, {X, 0.0, -Z},
            {0.0, Z, X}, {0.0, Z, -X}, {0.0, -Z, X}, {0.0, -Z, -X},
            {Z, X, 0.0}, {-Z, X, 0.0}, {Z, -X, 0.0}, {-Z, -X, 0.0}
    };
    static GLuint tindices[20][3] = {
            {0,4,1}, {0,9,4}, {9,5,4}, {4,5,8}, {4,8,1},
            {8,10,1}, {8,3,10}, {5,3,8}, {5,2,3}, {2,7,3},
            {7,10,3}, {7,6,10}, {7,11,6}, {11,0,6}, {0,1,6},
            {6,1,10}, {9,0,11}, {9,11,2}, {9,2,5}, {7,2,11} };

    for (int i = 0; i < 20; i++) {
        subdivide(&vertex_data[3 * 3 * (int) pow(4, depth) * i], vdata[tindices[i][0]], vdata[tindices[i][1]], vdata[tindices[i][2]], depth);
    }

    glGenBuffers(1, &vertices); // create buffer for these vertices
    glBindBuffer(GL_ARRAY_BUFFER, vertices);
    glBufferData(GL_ARRAY_BUFFER, 20 * 3 * 3 * (int) pow(4, depth) * sizeof(GLfloat), vertex_data, GL_STATIC_DRAW);

    GLfloat * color_data = new GLfloat[20 * 3 * 3 * (int) pow(4, depth)]; // TODO constant length array

    for (int i = 0; i < 20 * 3 * (int) pow(4, depth); i++) {
        color_data[3*i] = color[0];
        color_data[3*i + 1] = color[1];
        color_data[3*i + 2] = color[2];
    }

    glGenBuffers(1, &colors);
    glBindBuffer(GL_ARRAY_BUFFER, colors);
    glBufferData(GL_ARRAY_BUFFER, 20 * 3 * 3 * (int) pow(4, depth) * sizeof(GLfloat), color_data, GL_STATIC_DRAW);

    delete [] color_data;
    delete [] vertex_data;

    _initialized = true;
}

void Ball::draw() {
    glEnableVertexAttribArray(0);
    glBindBuffer(GL_ARRAY_BUFFER, vertices);

    glVertexAttribPointer(
            0,                  // attribute. No particular reason for 0, but must match the layout in the shader.
            3,                  // size
            GL_FLOAT,           // type
            GL_FALSE,           // normalized?
            0,                  // stride
            (void*)0            // array buffer offset
    );

    glEnableVertexAttribArray(1);
    glBindBuffer(GL_ARRAY_BUFFER, colors);
    glVertexAttribPointer(
            1,                                // attribute. No particular reason for 1, but must match the layout in the shader.
            3,                                // size
            GL_FLOAT,                         // type
            GL_FALSE,                         // normalized?
            0,                                // stride
            (void*)0                          // array buffer offset
    );

    // Draw the triangle !
    glDrawArrays(GL_TRIANGLES, 0, 20 * 3 * (int) pow(4, depth)); // 12*3 indices starting at 0 -> 12 triangles

    glDisableVertexAttribArray(1);
    glDisableVertexAttribArray(0);
}

#endif

#ifdef GRAPHICS
/*
 * Contact Plane Shader (Source: boxiXia)
 */
void ContactPlane::generateBuffers() {
    const int radius = 10; // radius [unit] of the plane
    // 10*10*4*6 = 2400
    // total 15*15*4*6=5400 points

    // define color
    glm::vec3 c1 = glm::vec3(0.729f, 0.78f, 0.655f);
    glm::vec3 c2 = glm::vec3(0.533f, 0.62f, 0.506f);
    // refer to: http://www.opengl-tutorial.org/intermediate-tutorials/tutorial-17-quaternions/
    std::vector<GLfloat> vertex_data;
    std::vector<GLfloat> color_data;

    GLfloat s = 0.1;// scale
    for (int i = -radius; i < radius; i++)
    {
        for (int j = -radius; j < radius; j++)
        {
            GLfloat x = i*s;
            GLfloat y = j*s;
            vertex_data.insert(vertex_data.end(), {
                    x,y,0,
                    x+s,y+s,0,
                    x+s,y,0,
                    x,y,0,
                    x,y+s,0,
                    x+s,y+s,0});//2 triangles of a quad
            // pick one color
            glm::vec3 c = (i + j) % 2 == 0? c1: c2;
            color_data.insert(color_data.end(), {
                    c[0],c[1],c[2],
                    c[0],c[1],c[2],
                    c[0],c[1],c[2],
                    c[0],c[1],c[2],
                    c[0],c[1],c[2],
                    c[0],c[1],c[2]});
        }
    }

    glm::vec3 glm_normal = glm::vec3(_normal[0], _normal[1], _normal[2]);
    auto quat_rot = glm::rotation(glm::vec3(0, 0, 1), glm_normal);

    glm::vec3 glm_offset = (float)_offset*glm_normal;

    #pragma omp parallel for
    for (size_t i = 0; i < vertex_data.size()/3; i++)
    {
        glm::vec3 v(vertex_data[3 * i], vertex_data[3 * i+1], vertex_data[3 * i+2]);
        v = glm::rotate(quat_rot, v) + glm_offset;
        vertex_data[3 * i] = v[0];
        vertex_data[3 * i+1] = v[1];
        vertex_data[3 * i+2] = v[2];
    }

    glGenBuffers(1, &vertices); // create buffer for these vertices
    glBindBuffer(GL_ARRAY_BUFFER, vertices);
    glBufferData(GL_ARRAY_BUFFER, sizeof(GLfloat)* vertex_data.size(), vertex_data.data(), GL_STATIC_DRAW);


    glGenBuffers(1, &colors);
    glBindBuffer(GL_ARRAY_BUFFER, colors);
    glBufferData(GL_ARRAY_BUFFER, sizeof(GLfloat) * color_data.size(), color_data.data(), GL_STATIC_DRAW);

    _initialized = true;
}

void ContactPlane::draw() {
    // 1st attribute buffer : vertices
    glEnableVertexAttribArray(0);
    glBindBuffer(GL_ARRAY_BUFFER, vertices);

    glVertexAttribPointer(
            0,                  // attribute. No particular reason for 0, but must match the layout in the shader.
            3,                  // size
            GL_FLOAT,           // type
            GL_FALSE,           // normalized?
            0,                  // stride
            (void*)0            // array buffer offset
    );

    glEnableVertexAttribArray(1);
    glBindBuffer(GL_ARRAY_BUFFER, colors);
    glVertexAttribPointer(
            1,                                // attribute. No particular reason for 1, but must match the layout in the shader.
            3,                                // size
            GL_FLOAT,                         // type
            GL_FALSE,                         // normalized?
            0,                                // stride
            (void*)0                          // array buffer offset
    );

    // Draw the triangle !
    glDrawArrays(GL_TRIANGLES, 0, 10*10*4*6); // number of vertices

    glDisableVertexAttribArray(1);
    glDisableVertexAttribArray(0);
}
#endif

} // namespace titan