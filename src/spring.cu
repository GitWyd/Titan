#include "hip/hip_runtime.h"
//
// Created by Jacob Austin on 5/17/18.
//
#include "spring.h"
#include <cmath>

namespace titan {


// The only parameter that is being changed for springs in CUDA is _rest
// Therefore, the update function is currently on updating the _rest parameter
void Spring::update(const CUDA_SPRING & s) {
        _rest = s._rest;
}

void Spring::defaultLength() { _rest = (_left -> pos - _right -> pos).norm() ; } // sets rest length

void Spring::setLeft(Mass * left) {
    if (_left) {
        _left -> decrementRefCount();
    }

    _left = left;
    _left -> ref_count++;

} // sets left mass (attaches spring to mass 1)

void Spring::setRight(Mass * right) {
    if (_right) {
        _right -> decrementRefCount();
    }

    _right = right;
    _right -> ref_count++;
}

CUDA_SPRING::CUDA_SPRING(const Spring & s) {
    _left = (s._left == nullptr) ? nullptr : s._left -> arrayptr;
    _right = (s._right == nullptr) ? nullptr : s. _right -> arrayptr;
    _k = s._k;
    _rest = s._rest;
    _type = s._type;
    _omega = s._omega;
    _damping = s._damping;
    // Actuator
    _l_max = s._l_max; // maximum actuator length
    _l_min = s._l_min; // minimum actuator length
    _rate = s._rate; // expansion rate [m/s]
}

CUDA_SPRING::CUDA_SPRING(const Spring & s, CUDA_MASS * left, CUDA_MASS * right) {
    _left = left;
    _right = right;
    _k = s._k;
    _rest = s._rest;
    _type = s._type;
    _omega = s._omega;
    _damping = s._damping;
    // Actuator
    _l_max = s._l_max; // maximum actuator length
    _l_min = s._l_min; // minimum actuator length
    _rate = s._rate; // expansion rate [m/s]
}

} // namespace titan